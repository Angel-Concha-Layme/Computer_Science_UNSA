#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include ""
#include <opencv2/core/core.hpp>
#include <opencv2/opencv.hpp>

#define CHANNELS 3

void Usage(char prog_name[]) 
{
    fprintf(stderr, "Usage: %s <image file path>\n", prog_name);
    exit(EXIT_FAILURE);
}

__global__      
void colorToGreyscaleConversion(unsigned char* Input_Image, unsigned char* Output_Image, int width, int height)
{
    int Col = blockIdx.x * blockDim.x + threadIdx.x;  
    int Row = blockIdx.y * blockDim.y + threadIdx.y; 

    if (Row < height && Col < width) { 
        int offset = Row * width + Col;
        
        int rgbOffset = offset * CHANNELS;
        

        unsigned char r = Input_Image[rgbOffset]; 

        unsigned char g = Input_Image[rgbOffset + 1];  

        unsigned char b = Input_Image[rgbOffset + 2];   


        Output_Image[offset] = 0.21f * r + 0.71f * g + 0.07f * b;
        
    }
}


int main(int argc, char** argv)
{
    if (argc != 2) {
        Usage(argv[0]);
    }

    const char* file_name = argv[1]; 
    int width, height, channels; 
    unsigned char* h_origImg, * h_resultImg;
    cv::Mat origImg = cv::imread(file_name); 

    width = origImg.cols; 
    height = origImg.rows; 
    channels = origImg.channels(); 

    printf("Image size = (%d x %d x %d)\n", width, height, channels); 
    assert(channels == CHANNELS); 

    cv::Mat half; 
    cv::resize(origImg, half, cv::Size(width / 2, height / 2));
    

    h_origImg = (unsigned char*)malloc(width * height * channels * sizeof(unsigned char));
   
    h_resultImg = (unsigned char*)malloc(width * height * sizeof(unsigned char));
    
    (void)memcpy(h_origImg, origImg.data, width * height * channels);
    

    unsigned char* d_origImg, * d_resultImg;
   
    hipMalloc((void**)&d_origImg, width * height * channels * sizeof(unsigned char));
    
    hipMalloc((void**)&d_resultImg, width * height * sizeof(unsigned char));
    

    hipMemcpy(d_origImg, h_origImg, width * height * channels * sizeof(unsigned char), hipMemcpyHostToDevice);
    

    const int block_size = 16; 
    dim3 threads(block_size, block_size); 
    dim3 grid(ceil(width / (double)threads.x), ceil(height / (double)threads.y));


    colorToGreyscaleConversion << <grid, threads >> > (d_origImg, d_resultImg, width, height);
    

    hipMemcpy(h_resultImg, d_resultImg, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost);


    cv::Mat resultImg(height, width, CV_8UC1); 
    memcpy(resultImg.data, h_resultImg, width * height); 

    hipFree(d_origImg); 
    hipFree(d_resultImg); 

    free(h_origImg); 
    free(h_resultImg); 

    cv::resize(resultImg, resultImg, cv::Size(width / 2, height / 2));


    cv::imwrite("grey.jpg", resultImg);

    return 0;
}



