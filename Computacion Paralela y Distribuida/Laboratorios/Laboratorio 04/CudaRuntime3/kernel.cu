#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include ""

#include <opencv2/core/core.hpp>
#include <opencv2/opencv.hpp>

#define CHANNELS 3
#define BLUR_SIZE 19

void Usage(char prog_name[]);
__global__
void blurKernel(unsigned char* Input_Image, unsigned char* Outpu_Image, int width, int height, int channel);

int main(int argc, char** argv)
{
    if (argc != 2) {
        Usage(argv[0]);
    }

    const char* file_name = argv[1];
    int width, height, channels;
    unsigned char* h_origImg, * h_resultImg;

    cv::Mat origImg = cv::imread(file_name);

    width = origImg.cols; 
    height = origImg.rows; 
    channels = origImg.channels(); 
    printf("Image size = (%d x %d x %d)\n", width, height, channels); 
    assert(channels == CHANNELS); 

    cv::Mat half; 
    cv::resize(origImg, half, cv::Size(width / 2, height / 2)); 
    cv::imshow("image", half); 
    cv::waitKey(0); 

    h_origImg = (unsigned char*)malloc(width * height * channels * sizeof(unsigned char));
    
    h_resultImg = (unsigned char*)malloc(width * height * channels * sizeof(unsigned char));
    
    (void)memcpy(h_origImg, origImg.data, width * height * channels);  


    unsigned char* d_origImg, * d_resultImg; 
    hipMalloc((void**)&d_origImg, width * height * channels * sizeof(unsigned char));
    
    hipMalloc((void**)&d_resultImg, width * height * channels * sizeof(unsigned char));
    


    hipMemcpy(d_origImg, h_origImg, width * height * channels * sizeof(unsigned char), hipMemcpyHostToDevice);
    


    const int block_size = 16; 
    dim3 threads(block_size, block_size, channels); 
    dim3 grid(ceil(height / (double)threads.x), ceil(width / (double)threads.y));
    
    blurKernel << <grid, threads >> > (d_origImg, d_resultImg, width, height, channels);
    


    hipMemcpy(h_resultImg, d_resultImg, width * height * channels * sizeof(unsigned char), hipMemcpyDeviceToHost);
    

    cv::Mat resultImg(height, width, CV_8UC3); 
    memcpy(resultImg.data, h_resultImg, width * height * channels); 


    
    hipFree(d_origImg);
    hipFree(d_resultImg);

    
    free(h_origImg);
    free(h_resultImg);


    cv::resize(resultImg, resultImg, cv::Size(width / 2, height / 2));
    cv::imwrite("Image_Blur.jpg", resultImg);

    return 0;
}

void Usage(char prog_name[])
{
    fprintf(stderr, "Usage: %s <image file path>\n", prog_name);
    exit(EXIT_FAILURE);
}

__global__
void blurKernel(unsigned char* Input_Image, unsigned char* Outpu_Image, int width, int height, int channel)
{
    int Plane = blockIdx.z * blockDim.z + threadIdx.z;
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;

    if (Row < height && Col < width && Plane < channel) {
        int pixelVal = 0;
        int pixelCnt = 0;

        for (int bRow = -BLUR_SIZE; bRow < BLUR_SIZE; bRow++) {
            for (int bCol = -BLUR_SIZE; bCol < BLUR_SIZE; bCol++) {
                int curRow = Row + bRow;
                int curCol = Col + bCol;

                if (curRow >= 0 && curRow < height && curCol >= 0 && curCol < width) {
                    pixelVal += Input_Image[(curRow * width + curCol) * channel + Plane];
                    pixelCnt++;
                }
            }
        }

        Outpu_Image[(Row * width + Col) * channel + Plane] = (unsigned char)(pixelVal / pixelCnt);
    }
}